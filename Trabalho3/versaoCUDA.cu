
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

// Tamanho do filtro, como definido na especificação: 5x5.
#define TAM_FILTRO 5

#define GRID 1
#define BLOCK 1024
#define TILE_WIDTH 28
#define BLOCK_WIDTH (TILE_WIDTH + (TAM_FILTRO - 1))

// Kernel de convolução
__global__ void convolucao(int *output, int *input, int width, int height){

        // Linha atual
        int idxY = blockIdx.y * TILE_WIDTH + threadIdx.y;

        // Coluna atual
        int idxX = blockIdx.x * TILE_WIDTH + threadIdx.x;

		// utilizacao de memoria compartilhada para diminuir o tempo de acesso aos pixels
        __shared__ int sharedMemory[BLOCK_WIDTH][BLOCK_WIDTH];

        int offset = TAM_FILTRO / 2;
        int sharedX = threadIdx.x, sharedY = threadIdx.y;
        int row_i = idxY - offset;
        int col_i = idxX - offset;

		// Loop que percorre todos os 3 canais de cor para realizar a convolucao
        for(int channelIndex = 0; channelIndex < 3; channelIndex++){

				// calculo de indice necessario, pois houve a transformacao da matriz da imagem em um vetor
                int index = (row_i * width + col_i) * 3 + channelIndex;

				// verifica se esta dentro dos limites
                if((row_i >= 0) && (row_i < height) &&
                        (col_i >= 0) && (col_i < width))
                                sharedMemory[sharedY][sharedX] = input[index];

                else
                        sharedMemory[sharedY][sharedX] = 0;

				// espera a matriz ser populada
                __syncthreads();

                int total = 0;

                // Verifica se valores de linha e coluna pertencem �|  matriz 
                if(sharedY < TILE_WIDTH && sharedX < TILE_WIDTH){
                        for(int i = 0; i < TAM_FILTRO; i++)
                                for(int j = 0; j < TAM_FILTRO; j++)
                                        total += sharedMemory[sharedY+i][sharedX+j];
                        // Média dos pixels
                        if(idxY < height && idxX < width)
                                output[(idxY * width + idxX) * 3 + channelIndex] = total / (TAM_FILTRO * TAM_FILTRO);

                        __syncthreads();
                }
        }
}

int main(int argc, char *argv[]){

        int totalX, totalY, N;

        FILE *arquivo;
        arquivo = fopen(argv[1], "r");

        if(arquivo == NULL){
                printf("Arquivo não pode ser encontrado!\n");
                exit(1);
        }

        // P3
        fseek(arquivo, 2, SEEK_SET);

        // tamanho da imagem
        fscanf(arquivo, "%d %d", &totalX, &totalY);
        N = totalX * totalY * 3;

        //declara matriz de entrada e final
        int *input, *outputFinal;

        input = (int *) malloc(sizeof(int) * N);
        outputFinal = (int *) malloc(sizeof(int) * N);

        //declara matrizes de entrada e saída (CUDA)
        int *dev_input, *dev_output;

        hipMalloc((void**) &dev_input, N * sizeof(int));
        hipMalloc((void**) &dev_output, N * sizeof(int));

        // 255
        fseek(arquivo, 4, SEEK_CUR);

        // carrega matriz
        //printf("%d %d\n", totalX, totalY);
        for(int i = 0; i < N; i++){
                fscanf(arquivo, "%d", &input[i]);
                //printf("input[%d]: %d\n", i, input[i]);
        }

        hipMemcpy(dev_input, input, N * sizeof(int), hipMemcpyHostToDevice);

		// dimensoes padrao de block e grid
        dim3 dimBlock(BLOCK_WIDTH, BLOCK_WIDTH);
        dim3 dimGrid((totalX-1) / TILE_WIDTH + 1, (totalY-1) / TILE_WIDTH + 1);

        convolucao<<<dimGrid, dimBlock>>>(dev_output, dev_input, totalX, totalY);

        hipMemcpy(outputFinal, dev_output, N * sizeof(int), hipMemcpyDeviceToHost);

        //Salva no arquivo de saida
        char str_final[100];

        strcpy(str_final, "out_cuda_");
        strcat(str_final, argv[1]);

        arquivo = fopen(str_final, "w");

        if(arquivo != NULL){
                fprintf(arquivo, "P3\n%d %d\n255\n", totalX, totalY);

                for(int i = 0; i < N; i++)
                        fprintf(arquivo, "%d\n", outputFinal[i]);

                fclose(arquivo);
        }

        else
                printf("Arquivo não pode ser criado!\n");

        free(input);
        free(outputFinal);
        hipFree(dev_input);
        hipFree(dev_output);

        return 0;
}
